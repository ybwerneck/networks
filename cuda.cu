#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>

// final time of simulation

// FHN cell model parameters
#define A 0.2
#define B 0.2
#define C 3.0
#define I_APP 1.0

// CUDA kernel to simulate FHN cell models
__global__ void fhn_kernel(float* ui, float* vi, float* u_solution, float* v_solution, float* t_solution, float DT, int NUM_CELLS, float T_FINAL, int rate, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // calculate global index of thread
    float u_i = ui[idx];
    float v_i = vi[idx];

    // check if index is within range of cells to simulate
    if (idx < NUM_CELLS) {
        int step = 0;
        for (int t_i = 0; t_i < N*rate; t_i++) {
            // update FHN model equations for current cell

            float u_new = u_i + DT * (10*(u_i*(u_i-0.4)*(1-u_i)-v_i));
            float v_new = v_i + DT * (0.5*(u_i*0.2-0.8*v_i));

            // store solution for current time step
            int ind = idx * N + step;

            if (step == 0 || (t_i) % rate == 0) {
                u_solution[ind] = u_i;
                v_solution[ind] = v_i;
                t_solution[ind] = t_i * DT;
                step++;
            }
            u_i = u_new;
            v_i = v_new;
        }
    }
}

int main(int argc, char* argv[]) {

    float DT = 0.1;
    float T_FINAL =100;
    int rate = 2;

    if (argc != 4) {
        printf("Usage: program_name arg1 arg2\n");
  
    }
    else {
        DT = atof(argv[2]);
        T_FINAL = atof(argv[1]);
        rate = atoi(argv[3]);
    }
    int N=T_FINAL / (DT * rate);
    if(T_FINAL / (DT * rate)>int(T_FINAL / (DT * rate)))
        N++;


    int NUM_CELLS = 0;
    std::ifstream file("u.csv");
    std::string row;
    while (std::getline(file, row)) {
        NUM_CELLS++;
    }

    printf("%d", NUM_CELLS);


    // allocate memory on host for FHN model variables
    float* u_host = (float*)malloc(sizeof(float) * NUM_CELLS);
    float* v_host = (float*)malloc(sizeof(float) * NUM_CELLS);
    float* t_host = (float*)malloc(sizeof(float) * NUM_CELLS);


    float* u_solution = (float*)malloc(sizeof(float) * NUM_CELLS * N);
    float* v_solution = (float*)malloc(sizeof(float) * NUM_CELLS * N);
    float* t_solution = (float*)malloc(sizeof(float) * NUM_CELLS * N);


    // initialize FHN model variables on host
    for (int i = 0; i < NUM_CELLS; i++) {
        t_host[i] = 0.0;
        v_host[i] = 0.0;
    }

    std::ifstream file2("u.csv");
    std::string line;
    int i = 0;
    while (std::getline(file2, line) && i < NUM_CELLS) {
        std::stringstream ss(line);
        std::string cell;
        while (std::getline(ss, cell, ',') && i < NUM_CELLS) {
            u_host[i] = std::stof(cell);
            i++;
        }
    }
    file.close();

    // allocate memory on device for FHN model variables
    float* u_dev, * v_dev, * t_dev;
    hipMalloc((void**)&u_dev, sizeof(float) * NUM_CELLS);
    hipMalloc((void**)&v_dev, sizeof(float) * NUM_CELLS);
    hipMalloc((void**)&t_dev, sizeof(float) * NUM_CELLS);


    float* u_solution_dev, * v_solution_dev, * t_solution_dev;
    hipMalloc((void**)&u_solution_dev, sizeof(float) * NUM_CELLS * N);
    hipMalloc((void**)&v_solution_dev, sizeof(float) * NUM_CELLS * N);
    hipMalloc((void**)&t_solution_dev, sizeof(float) * NUM_CELLS * N);

    // copy FHN model variables from host to device
    hipMemcpy(u_dev, u_host, sizeof(float) * NUM_CELLS, hipMemcpyHostToDevice);
    hipMemcpy(v_dev, v_host, sizeof(float) * NUM_CELLS, hipMemcpyHostToDevice);
    hipMemcpy(t_dev, t_host, sizeof(float) * NUM_CELLS, hipMemcpyHostToDevice);

    // calculate number of CUDA threads and blocks to use
    int threads_per_block = 256;
    int blocks_per_grid = (NUM_CELLS + threads_per_block - 1) / threads_per_block;

    // simulate FHN cell models on device using CUDA kernel
    fhn_kernel << <blocks_per_grid, threads_per_block >> > (u_dev, v_dev, u_solution_dev, v_solution_dev, t_solution_dev, DT, NUM_CELLS, T_FINAL, rate,N);

    // copy FHN model variables from device to host
    hipMemcpy(u_solution, u_solution_dev, sizeof(float) * NUM_CELLS * N, hipMemcpyDeviceToHost);
    hipMemcpy(v_solution, v_solution_dev, sizeof(float) * NUM_CELLS * N, hipMemcpyDeviceToHost);
    hipMemcpy(t_solution, t_solution_dev, sizeof(float) * NUM_CELLS * N, hipMemcpyDeviceToHost);

    FILE* u_fp, * v_fp, * t_fp;
    u_fp = fopen("outputs/u.csv", "w");
    v_fp = fopen("outputs/v.csv", "w");
    t_fp = fopen("outputs/t.csv", "w");

    for (int i = 0; i < NUM_CELLS; i++) {
        fprintf(u_fp, "%f", u_host[i]);
        fprintf(v_fp, "%f", v_host[i]);

        for (int j = 1; j < N; j++) {
            int ind = i * N + j;
            // update FHN model equations for current cell
            float U = u_solution[ind];
            float V = v_solution[ind];
            // print updated values to CSV files
            fprintf(u_fp, ",%f", U);
            fprintf(v_fp, ",%f", V);

        }
        fprintf(u_fp, "\n");
        fprintf(v_fp, "\n");
    }
    fclose(u_fp);
    fclose(v_fp);
    fprintf(t_fp, "%f", 0.0);

    for (int i = 0; i < N; i++) {
        fprintf(t_fp, ", %f ", t_solution[i]);
    }

    fclose(t_fp);

    // free memory
    free(u_host);
    free(v_host);
    free(t_host);
    hipFree(u_dev);
    hipFree(v_dev);
    hipFree(t_dev);

    return 0;
}
